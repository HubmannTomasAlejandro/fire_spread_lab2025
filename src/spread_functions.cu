#include "hip/hip_runtime.h"
#include "spread_functions.hpp"

#define _USE_MATH_DEFINES
#include <cmath>
#include <vector>
#include <cstdint>
#include <iostream>
#include <cstddef>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <array>


#include "fires.hpp"
#include "landscape.hpp"
#include "constants.hpp"

__constant__ float DEV_ANGLES[8];
__constant__ int DEV_MOVES[8][2];


CUDA_CALLABLE float spread_probability_scalar(
    const Cell& burning,
    const Cell& neighbor,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float angle,
    float upper_limit,
    const SimulationParams& params
) {
    std::array<float,4> veg_pred = {0.f,0.f,0.f,0.f};
    veg_pred[static_cast<int>(VegetationType::SUBALPINE)] = params.subalpine_pred;
    veg_pred[static_cast<int>(VegetationType::WET)] = params.wet_pred;
    veg_pred[static_cast<int>(VegetationType::DRY)] = params.dry_pred;

    if (!neighbor.burnable) return 0.0f;

    float slope = (neighbor.elevation - burning.elevation) / distance;
    float slope_term = sin(atan(slope));
    float wind_term = cos(angle - burning.wind_direction);
    float elev_term = (neighbor.elevation - elevation_mean) / elevation_sd;

    float linear_pred = params.independent_pred;
    linear_pred += veg_pred[static_cast<int>(neighbor.vegetation_type)];
    linear_pred += params.fwi_pred * neighbor.fwi;
    linear_pred += params.aspect_pred * neighbor.aspect;
    linear_pred += params.wind_pred * wind_term;
    linear_pred += params.elevation_pred * elev_term;
    linear_pred += params.slope_pred * slope_term;

    return upper_limit / (1.0f + exp(-linear_pred));
}

__global__ void fire_spread_kernel(
    const Cell* landscape,
    unsigned int* burning_state,
    hiprandStateXORWOW_t* rng_states,  // Added pre-initialized RNG states
    size_t width, size_t height,
    float distance, float elevation_mean, float elevation_sd,
    float upper_limit, const SimulationParams params,
    unsigned int current_iteration,
    bool* active_flag
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    size_t idx = y * width + x;

    // Only process cells that burned in previous iteration
    if (burning_state[idx] != current_iteration - 1) return;

    Cell burning_cell = landscape[idx];
    bool thread_active = false;

    //unsigned long seed = (123456 * current_iteration) % idx;
    //unsigned long seed = 123456 + current_iteration * 31 + idx;

    hiprandStateXORWOW_t rng_state = rng_states[idx];
    //hiprand_init(seed + idx, idx, 0, &rng_state);

    for (int i = 0; i < 8; i++) {
        int nx = x + DEV_MOVES[i][0];
        int ny = y + DEV_MOVES[i][1];
        if (nx < 0 || nx >= width || ny < 0 || ny >= height) continue;

        size_t neighbor_idx = ny * width + nx;
        if (burning_state[neighbor_idx] != 0) continue; // Skip already burned

        Cell neighbor_cell = landscape[neighbor_idx];
        if (!neighbor_cell.burnable) continue;

        float prob = spread_probability_scalar(
            burning_cell, neighbor_cell, distance,
            elevation_mean, elevation_sd, DEV_ANGLES[i],
            upper_limit, params
        );

        float rand_val = hiprand_uniform(&rng_state);

        if (rand_val < prob) {
            burning_state[neighbor_idx] = current_iteration;
            thread_active = true;
        }
    }

    rng_states[idx] = rng_state; // Save the updated RNG state back to device memory

    if (thread_active) {
        *active_flag = true;
    }
}

Fire simulate_fire(
    const Landscape& landscape,
    const Cell* d_landscape,
    const std::vector<IgnitionPair>& ignition_cells,
    unsigned int* d_burning_state,
    hiprandStateXORWOW_t* d_rng_states,
    SimulationParams params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float upper_limit
) {
    size_t width = landscape.width;
    size_t height = landscape.height;
    size_t num_cells = width * height;

    Fire result{
        landscape.width,
        landscape.height,
        d_burning_state,
        std::vector<IgnitionPair>(),
        std::vector<size_t>()
    };

    hipMemset(d_burning_state, 0, num_cells * sizeof(unsigned int));

    unsigned int value = 1;
    for (const auto& cell : ignition_cells) {
        size_t idx = cell.second * width + cell.first;
        hipMemcpy(d_burning_state + idx, &value, sizeof(unsigned int), hipMemcpyHostToDevice);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(DEV_ANGLES), ANGLES, 8 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(DEV_MOVES), MOVES, 8 * 2 * sizeof(int));

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                 (height + blockSize.y - 1) / blockSize.y);

    bool* d_active_flag;
    hipMalloc(&d_active_flag, sizeof(bool));

    unsigned int current_iteration = 2;
    bool h_active = true;


    while (h_active) {
        h_active = false;
        hipMemset(d_active_flag, 0, sizeof(bool));

        fire_spread_kernel<<<gridSize, blockSize>>>(
            d_landscape, d_burning_state, d_rng_states,
            width, height,
            distance, elevation_mean, elevation_sd, upper_limit,
            params, current_iteration,
            d_active_flag
        );

        hipDeviceSynchronize();
        hipMemcpy(&h_active, d_active_flag, sizeof(bool), hipMemcpyDeviceToHost);

        current_iteration++;
    }

    unsigned int* h_burned_layer = new unsigned int[num_cells];

    // Copy from device to host
    hipMemcpy(h_burned_layer, d_burning_state,
            num_cells * sizeof(unsigned int),
            hipMemcpyDeviceToHost);

    result.width = width;
    result.height = height;
    result.burned_layer = h_burned_layer;

    hipFree(d_active_flag);

    return result;
}