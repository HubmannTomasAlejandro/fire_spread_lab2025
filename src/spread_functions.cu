#include "hip/hip_runtime.h"
#include "spread_functions.hpp"

#define _USE_MATH_DEFINES
#include <cmath>
#include <cstdint>
#include <vector>
#include <iostream>
#include <bitset>

#include "fires.hpp"
#include "landscape.hpp"
#include "constants.hpp"

class XorShift32 {
private:
    uint32_t state;

public:
    __host__ __device__
    explicit XorShift32(uint32_t seed = 12345) {
        state = seed ? seed : 2463534242U; // Evita estado cero
    }

    __host__ __device__
    float nextFloat() {
        state ^= state << 13;
        state ^= state >> 17;
        state ^= state << 5;
        return static_cast<float>(state) / static_cast<float>(UINT32_MAX);
    }
};

void inline spread_probability(
  const Landscape& landscape,
  const Cell& burning,
  const int neighbours[2][8], 
  SimulationParams params,
  float distance,
  float elevation_mean,
  float elevation_sd,
  float* probs,
  std::bitset<8>& burnable_cell,
  float upper_limit = 1.0f
) {
  IgnitionPair neighbour;
  for (size_t i = 0; i < 8; i++) {
    neighbour.first = neighbours[0][i];
    neighbour.second = neighbours[1][i];

    float slope_term = sinf(atanf((landscape.elevations[neighbour] - burning.elevation) / distance);
    float wind_term = cosf(ANGLES[i] - burning.wind_direction);
    float elev_term = (landscape.elevations[neighbour] - elevation_mean) / elevation_sd;

    float linpred = params.independent_pred;

    if (landscape.vegetation_types[neighbour] == SUBALPINE) {
        linpred += params.subalpine_pred;
    } else if (landscape.vegetation_types[neighbour] == WET) {
        linpred += params.wet_pred;
    } else if (landscape.vegetation_types[neighbour] == DRY) {
        linpred += params.dry_pred;
    }

    linpred += params.fwi_pred * landscape.fwis[neighbour];
    linpred += params.aspect_pred * landscape.aspects[neighbour];
    linpred += wind_term * params.wind_pred +
               elev_term * params.elevation_pred +
               slope_term * params.slope_pred;

    probs[i] = (landscape.vegetation_types[neighbour] == NONE || !burnable_cell[i])
               ? 0.0f
               : upper_limit / (1.0f + expf(-linpred));
  }
}

Fire simulate_fire(
    const Landscape& landscape, 
    const std::vector<IgnitionPair>& ignition_cells,
    SimulationParams params, 
    float distance, 
    float elevation_mean, 
    float elevation_sd,
    float upper_limit = 1.0
) {
  size_t n_row = landscape.height;
  size_t n_col = landscape.width;

  std::vector<IgnitionPair> burned_ids;
  burned_ids.insert(burned_ids.end(), ignition_cells.begin(), ignition_cells.end());

  std::vector<size_t> burned_ids_steps;
  burned_ids_steps.push_back(ignition_cells.size());

  size_t start = 0;
  size_t end = ignition_cells.size();
  size_t burning_size = end - start;

  Matrix<bool> burned_bin(n_col, n_row);
  for (const auto& cell : ignition_cells) {
    burned_bin[cell] = true;
  }

  int neighbours_coords[2][8];
  float probs[8];
  XorShift32 rng(12345); // Inicializado con semilla fija

  while (burning_size > 0) {
    size_t end_forward = end;

    for (size_t b = start; b < end; b++) {
      const IgnitionPair& burning_id = burned_ids[b];
      size_t burning_cell_0 = burning_id.first;
      size_t burning_cell_1 = burning_id.second;

      std::bitset<8> burnable_cell;
      const Cell& burning_cell = landscape[burning_id];

      for (size_t i = 0; i < 8; i++) {
        neighbours_coords[0][i] = burning_cell_0 + MOVES[i][0];
        neighbours_coords[1][i] = burning_cell_1 + MOVES[i][1];

        bool out_of_range = 
          neighbours_coords[0][i] >= n_col || 
          neighbours_coords[1][i] >= n_row;

        if (out_of_range) {
          burnable_cell[i] = false;
        } else {
          IgnitionPair neighbour(neighbours_coords[0][i], neighbours_coords[1][i]);
          burnable_cell[i] = !burned_bin[neighbour] && landscape.burnables[neighbour];
        }
      }

      if (burnable_cell.none()) continue;

      spread_probability(
        landscape, burning_cell, neighbours_coords, params, distance,
        elevation_mean, elevation_sd, probs, burnable_cell, upper_limit
      );

      for (size_t i = 0; i < 8; i++) {
        if (!burnable_cell[i]) continue;

        float rand_val = rng.nextFloat();
        if (rand_val < probs[i]) {
          IgnitionPair new_burn(neighbours_coords[0][i], neighbours_coords[1][i]);
          burned_ids.push_back(new_burn);
          burned_bin[new_burn] = true;
          end_forward++;
        }
      }
    }

    start = end;
    end = end_forward;
    burning_size = end - start;
    burned_ids_steps.push_back(end);
  }

  return {n_col, n_row, burned_bin, burned_ids, burned_ids_steps};
}