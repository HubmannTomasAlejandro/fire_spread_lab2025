#include "hip/hip_runtime.h"
#include "spread_functions.hpp"

#define _USE_MATH_DEFINES
#include <cmath>
#include <vector>
#include <cstdint>
#include <iostream>
#include <cstddef>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <array>


#include "fires.hpp"
#include "landscape.hpp"
#include "constants.hpp"

__constant__ float DEV_ANGLES[8];
__constant__ int DEV_MOVES[8][2];


CUDA_CALLABLE float spread_probability_scalar(
    const Cell& burning,
    const Cell& neighbor,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float angle,
    float upper_limit,
    const SimulationParams& params
) {
    std::array<float,4> veg_pred = {0.f,0.f,0.f,0.f};
    veg_pred[static_cast<int>(VegetationType::SUBALPINE)] = params.subalpine_pred;
    veg_pred[static_cast<int>(VegetationType::WET)] = params.wet_pred;
    veg_pred[static_cast<int>(VegetationType::DRY)] = params.dry_pred;

    if (!neighbor.burnable) return 0.0f;

    float slope = (neighbor.elevation - burning.elevation) / distance;
    float slope_term = sin(atan(slope));
    float wind_term = cos(angle - burning.wind_direction);
    float elev_term = (neighbor.elevation - elevation_mean) / elevation_sd;

    float linear_pred = params.independent_pred;
    linear_pred += veg_pred[static_cast<int>(neighbor.vegetation_type)];
    linear_pred += params.fwi_pred * neighbor.fwi;
    linear_pred += params.aspect_pred * neighbor.aspect;
    linear_pred += params.wind_pred * wind_term;
    linear_pred += params.elevation_pred * elev_term;
    linear_pred += params.slope_pred * slope_term;

    return upper_limit / (1.0f + exp(-linear_pred));
}

__global__ void fire_spread_kernel(
    const Cell* landscape,
    unsigned int* burning_state,
    size_t width, size_t height,
    float distance, float elevation_mean, float elevation_sd,
    float upper_limit, const SimulationParams params,
    unsigned int current_iteration,
    bool* active_flag
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    size_t idx = y * width + x;

    // Only process cells that burned in previous iteration
    if (burning_state[idx] != current_iteration - 1) return;

    unsigned long seed =
        (blockIdx.x * 2654435761) ^
        (threadIdx.x * 2246822519) ^
        (current_iteration * 3266489917) ^
        (clock64() % 7919);
    hiprandStatePhilox4_32_10_t rng_state;
    hiprand_init(seed, idx, 0, &rng_state);

    Cell burning_cell = landscape[idx];
    bool thread_active = false;

    for (int i = 0; i < 8; i++) {
        int nx = x + DEV_MOVES[i][0];
        int ny = y + DEV_MOVES[i][1];
        if (nx < 0 || nx >= width || ny < 0 || ny >= height) continue;

        size_t neighbor_idx = ny * width + nx;
        if (burning_state[neighbor_idx] != 0) continue; // Skip already burned

        Cell neighbor_cell = landscape[neighbor_idx];
        if (!neighbor_cell.burnable) continue;

        float prob = spread_probability_scalar(
            burning_cell, neighbor_cell, distance,
            elevation_mean, elevation_sd, DEV_ANGLES[i],
            upper_limit, params
        );

        float rand_val = hiprand_uniform(&rng_state);

        if (rand_val < prob) {
            burning_state[neighbor_idx] = current_iteration;
            thread_active = true;
        }
    }

    if (thread_active) {
        *active_flag = true;
    }
}

__global__ void setup_rng_kernel(hiprandState* state, unsigned long seed, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}

void simulate_fire_cuda(
    const Landscape& landscape,
    const std::vector<IgnitionPair>& ignition_cells,
    SimulationParams params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float upper_limit,
    Fire& result
) {
    size_t width = landscape.width;
    size_t height = landscape.height;
    size_t num_cells = width * height;

    Cell* d_landscape;
    hipMalloc(&d_landscape, num_cells * sizeof(Cell));
    hipMemcpy(d_landscape, landscape.cells, num_cells * sizeof(Cell), hipMemcpyHostToDevice);

    unsigned int* d_burning_state;
    hipMalloc(&d_burning_state, num_cells * sizeof(unsigned int));
    hipMemset(d_burning_state, 0, num_cells * sizeof(unsigned int));

    unsigned int value = 1;
    for (const auto& cell : ignition_cells) {
        size_t idx = cell.second * width + cell.first;
        hipMemcpy(d_burning_state + idx, &value, sizeof(unsigned int), hipMemcpyHostToDevice);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(DEV_ANGLES), ANGLES, 8 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(DEV_MOVES), MOVES, 8 * 2 * sizeof(int));


    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                 (height + blockSize.y - 1) / blockSize.y);

    bool* d_active_flag;
    hipMalloc(&d_active_flag, sizeof(bool));

    unsigned int current_iteration = 2;
    bool h_active = true;

    for (int i = 0; i < 10; i++) {
        // Copy just this cell's data
        unsigned int state;
        Cell cell;
        hipMemcpy(&state, d_burning_state + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(&cell, d_landscape + i, sizeof(Cell), hipMemcpyDeviceToHost);


        if (cell.burnable != landscape.cells[i].burnable ) {
            printf("Cells in device memory:\n");
            printf("Cell %d - Burnable: %d, Vegetation: %d, State: %u\n",
                i, cell.burnable, static_cast<int>(cell.vegetation_type), state);
            printf("Cells in landscape:\n");
            printf("Cell %d - Burnable: %d, Vegetation: %d\n",
                i, landscape.cells[i].burnable, static_cast<int>(landscape.cells[i].vegetation_type));
        }


    }

    while (h_active) {
        h_active = false;
        hipMemset(d_active_flag, 0, sizeof(bool));

        fire_spread_kernel<<<gridSize, blockSize>>>(
            d_landscape, d_burning_state, width, height,
            distance, elevation_mean, elevation_sd, upper_limit,
            params, current_iteration,
            d_active_flag
        );

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel error: %s\n", hipGetErrorString(err));
        }

        hipDeviceSynchronize();
        hipMemcpy(&h_active, d_active_flag, sizeof(bool), hipMemcpyDeviceToHost);

        current_iteration++;
    }


    std::vector<unsigned int> temp_burned_layer(num_cells);
    hipMemcpy(temp_burned_layer.data(), d_burning_state,
            num_cells * sizeof(unsigned int), hipMemcpyDeviceToHost);



    std::vector<IgnitionPair> burned_ids_aux;
    std::vector<size_t> burned_ids_steps_aux;

    // Convert to Matrix<bool>
    Matrix<bool> burned_layer(width, height);
    unsigned int burned_amount = 0;
    for (size_t i = 0; i < num_cells; i++) {
        burned_layer.data()[i] = (temp_burned_layer[i] > 0);
        burned_amount += int(burned_layer.data()[i]);
    }
    result.width = width;
    result.height = height;
    result.burned_layer = burned_layer;

    hipFree(d_landscape);
    hipFree(d_burning_state);
    hipFree(d_active_flag);
}

Fire simulate_fire(
    const Landscape& landscape,
    const std::vector<std::pair<size_t, size_t>>& ignition_cells,
    SimulationParams params,
    float distance,
    float elevation_mean,
    float elevation_sd,
    float upper_limit
) {
    std::vector<IgnitionPair> ignition;
    ignition.reserve(ignition_cells.size());
    for (const auto& p : ignition_cells) {
        ignition.emplace_back(p.first, p.second);
    }

    Fire result{
        landscape.width,
        landscape.height,
        Matrix<bool>(landscape.width, landscape.height),
        std::vector<IgnitionPair>(),
        std::vector<size_t>()
    };

    simulate_fire_cuda(
        landscape, ignition, params,
        distance, elevation_mean, elevation_sd,
        upper_limit, result
    );

    unsigned int burned_amount = 0;
    for (size_t i = 0; i < landscape.width * landscape.height; i++) {
        burned_amount += int(result.burned_layer.data()[i]);
    }


    return result;
}